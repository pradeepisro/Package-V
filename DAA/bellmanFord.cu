#include "bellmanFord.hpp"

cudaError_h::cudaError_h(hipError_t err){
    if(err != hipSuccess)
        printf("%s in file %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
}

bellmanFord::bellmanFord(std::string filename){
    fin.open(filename.c_str());
    if(fin.fail()){
        std::cout << "unable to open the file!\n";
        std::exit(0);
    }

    fin >> this->nNodes;
    fin >> this->nEdges;

    this->h_distance = new int[nNodes];
    this->h_pi = new int[nNodes];
    this->h_index = new int[nNodes];
    this->h_edge = new int[nEdges];
    this->h_weight = new int[nEdges];

    int current_node = 0, current_count = 0;
    h_index[0] = 0;
    for(int i = 0, j = 0; i < nNodes; i++){
        fin >> current_node;
        fin >> current_count;
        this->h_pi[i] = -1;
        this->h_distance[i] = 100000;
        for(j = 0; j < current_count; j++){
            fin >> h_edge[h_index[current_node] + j];
            fin >> h_weight[h_index[current_node] + j];
        }

        this->h_index[current_node + 1] = h_index[current_node] + current_count;
    }
}

int bellmanFord::gpuInit(){

    hipDeviceProp_t prop;
    int dev_count = 0;
    hipGetDeviceCount(&dev_count);
    std::cout << "Number of Devices detected: " << dev_count << std::endl;
    for(int i = 0; i < dev_count; i++){
        cudaError_h(hipGetDeviceProperties(&prop, 0));
        std::cout << "---------------device detected----------------\n";
        std::cout << "Device name: " << prop.name << std::endl;
        std::cout << "Max Threads per count: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "Compute Capability: " << prop.major << std::endl;
        std::cout << "----------------------------------------------\n";
    }

    cudaError_h(hipMalloc((void**)&this->d_index, this->nNodes * sizeof(int)));
    cudaError_h(hipMemcpy(this->d_index, this->h_index, this->nNodes * sizeof(int), hipMemcpyHostToDevice));

    cudaError_h(hipMalloc((void**)&this->d_distance, nNodes * sizeof(int)));
    cudaError_h(hipMemcpy(this->d_distance, this->h_distance, nNodes * sizeof(int), hipMemcpyHostToDevice));

    cudaError_h(hipMalloc((void**)&this->d_pi, this->nNodes * sizeof(int)));
    cudaError_h(hipMemcpy(this->d_pi, this->h_pi, this->nNodes * sizeof(int), hipMemcpyHostToDevice));

    cudaError_h(hipMalloc((void**)&this->d_edge, this->nEdges * sizeof(int)));
    cudaError_h(hipMemcpy(this->d_edge, this->h_edge, this->nEdges * sizeof(int), hipMemcpyHostToDevice));

    cudaError_h(hipMalloc((void**)&this->d_weight, this->nEdges * sizeof(int)));
    cudaError_h(hipMemcpy(this->d_weight, this->h_weight, this->nEdges * sizeof(int), hipMemcpyHostToDevice));

    return 0;
}

void bellmanFord::displayPrecedence(){
    std::cout << "The precedence of the given graph:\n";
    for(int i = 0; i < this->nNodes; i++){
        std::cout << i << " " << this->h_pi[i] << "\n";
    }

}

void bellmanFord::shortestPath(int source){
    h_distance[source] = 0;
    this->gpuInit();
    
}