#include "hip/hip_runtime.h"
#include "bellmanFord.hpp"

__device__ globalParams gpuParams;

cudaError_h::cudaError_h(hipError_t err){
    if(err != hipSuccess)
        printf("%s in file %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
}

bellmanFord::bellmanFord(std::string filename){
    fin.open(filename.c_str());
    if(fin.fail() || filename == ""){
        std::cout << "unable to open the file!\n";
        std::exit(0);
    }

    fin >> this->nNodes;
    fin >> this->nEdges;

    this->h_distance = new int[nNodes];
    this->h_pi = new int[nNodes];
    this->h_index = new int[nNodes + 1];
    this->h_queue_1 = new bool[nNodes];
    this->h_queue_2 = new bool[nNodes];

    this->h_edge = new int[nEdges];
    this->h_weight = new int[nEdges];
    this->h_iteration = new int[nNodes];

    int current_node = 0, current_count = 0;
    h_index[0] = 0;
    for(int i = 0, j = 0; i < nNodes; i++){
        fin >> current_node;
        fin >> current_count;
        this->h_queue_1[i] = false;
        this->h_queue_2[i] = false;
        this->h_pi[i] = -1;
        this->h_iteration[i] = 0;
        this->h_distance[i] = 100000;
        for(j = 0; j < current_count; j++){
            fin >> h_edge[h_index[current_node] + j];
            fin >> h_weight[h_index[current_node] + j];
        }
        this->h_index[current_node + 1] = h_index[current_node] + current_count;
    }
    this->h_index[nNodes] = this->nEdges;
}

int bellmanFord::gpuInit(int source){

    hipDeviceProp_t prop;
    int dev_count = 0;
    hipGetDeviceCount(&dev_count);
    std::cout << "Number of Devices detected: " << dev_count << std::endl;
    for(int i = 0; i < dev_count; i++){
        cudaError_h(hipGetDeviceProperties(&prop, 0));
        std::cout << "---------------device detected----------------\n";
        std::cout << "Device name: " << prop.name << std::endl;
        std::cout << "Max Threads per count: " << prop.maxThreadsPerBlock << std::endl;
        std::cout << "Compute Capability: " << prop.major << std::endl;
        std::cout << "----------------------------------------------\n";
    }

    h_queue_1[source] = true;
    h_distance[source] = 0;

    cudaError_h(hipMalloc((void**)&this->d_index, (this->nNodes + 1) * sizeof(int)));
    cudaError_h(hipMemcpy(this->d_index, this->h_index, (this->nNodes + 1) * sizeof(int), hipMemcpyHostToDevice));

    cudaError_h(hipMalloc((void**)&this->d_distance, nNodes * sizeof(int)));
    cudaError_h(hipMemcpy(this->d_distance, this->h_distance, nNodes * sizeof(int), hipMemcpyHostToDevice));

    cudaError_h(hipMalloc((void**)&this->d_pi, this->nNodes * sizeof(int)));
    cudaError_h(hipMemcpy(this->d_pi, this->h_pi, this->nNodes * sizeof(int), hipMemcpyHostToDevice));

    cudaError_h(hipMalloc((void**)&this->d_edge, this->nEdges * sizeof(int)));
    cudaError_h(hipMemcpy(this->d_edge, this->h_edge, this->nEdges * sizeof(int), hipMemcpyHostToDevice));

    cudaError_h(hipMalloc((void**)&this->d_weight, this->nEdges * sizeof(int)));
    cudaError_h(hipMemcpy(this->d_weight, this->h_weight, this->nEdges * sizeof(int), hipMemcpyHostToDevice));

    cudaError_h(hipMalloc((void**)&this->d_queue_1, this->nEdges * sizeof(bool)));
    cudaError_h(hipMemcpy(this->d_queue_1, this->h_queue_1, this->nNodes * sizeof(bool), hipMemcpyHostToDevice));

    cudaError_h(hipMalloc((void**)&this->d_queue_2, this->nEdges * sizeof(bool)));
    cudaError_h(hipMemcpy(this->d_queue_2, this->h_queue_2, this->nNodes * sizeof(bool), hipMemcpyHostToDevice));

    cudaError_h(hipMalloc((void**)&this->d_iteration, this->nNodes * sizeof(int)));
    cudaError_h(hipMemcpy(this->d_iteration, this->h_iteration, this->nNodes * sizeof(int), hipMemcpyHostToDevice));


    globalParams hostParams;

    hostParams.nNodes = this->nNodes;
    hostParams.nEdges = this->nEdges;
    hostParams.index = d_index;
    hostParams.edge = d_edge;
    hostParams.weight = d_weight;
    hostParams.f1 = d_queue_1;
    hostParams.f2 = d_queue_2;
    hostParams.iteration = d_iteration;
    hostParams.distance = d_distance;
    hostParams.pi = d_pi;

    hipMemcpyToSymbol(HIP_SYMBOL(gpuParams), &hostParams, sizeof(globalParams));

    std::cout << "GPU init: Done initializing the device(s)!" << std::endl;

    return 0;
}

void bellmanFord::displayPrecedence(){
    std::cout << "The precedence of the given graph:\n";
    for(int i = 0; i < this->nNodes; i++){
        std::cout << i << " " << this->h_pi[i] << " " << h_distance[i] << "\n";
    }
}

__global__
void computeShortestPath(){
    printf("kernel executing!\n");
    int tid = threadIdx.x + blockDim.x * blockIdx.x;
    if(tid > gpuParams.nNodes){
        return;
    }

    __shared__ bool toggle;
    __shared__ int iter;
    int count = 0;
    if(tid == 0){
        toggle = false;
        iter = 0;
        count = 0;
    }
    __syncthreads();
    while(iter < 7){
        if(gpuParams.f1[tid]){
            gpuParams.f1[tid] = false;
            count = gpuParams.index[tid + 1] - gpuParams.index[tid];
            printf("tid=%d count=%d\n", tid, count);
            relax<<<1, count>>>(tid, count);
            hipDeviceSynchronize();
        }
        __syncthreads();
        if(tid == 0){
            bool *temp = gpuParams.f1;
            gpuParams.f1 = gpuParams.f2;
            gpuParams.f2 = temp;
            iter++;
        }
        __syncthreads();
    }

    if(tid == 0){
        for(int i = 0; i < gpuParams.nNodes; i++){
            printf("the distance: %d %d\n", i, gpuParams.distance[i]);
        }
    }

}

void bellmanFord::shortestPath(int source){
    h_distance[source] = 0;
    this->gpuInit(source);
    hipError_t kernel_launch_error;
    computeShortestPath<<<1, nNodes>>>();
    hipDeviceSynchronize();
    kernel_launch_error = hipGetLastError();
    if(kernel_launch_error != hipSuccess){
        std::cout << hipGetErrorString(kernel_launch_error) << std::endl;
    }
    std::cout << "Exiting bellmanFord!\n";
    hipMemcpy(this->h_distance, gpuParams.distance, this->nNodes * sizeof(int), hipMemcpyDeviceToHost);
}

bellmanFord::~bellmanFord(){
    free(h_index);
    free(h_edge);
    free(h_iteration);
    free(h_weight);
    free(h_distance);
    free(h_pi);
    free(h_queue_1);
    free(h_queue_2);

    hipFree(d_index);
    hipFree(d_edge);
    hipFree(d_weight);
    hipFree(d_iteration);
    hipFree(d_distance);
    hipFree(d_pi);
    hipFree(d_iteration);
    hipFree(d_queue_1);
    hipFree(d_queue_2);
}